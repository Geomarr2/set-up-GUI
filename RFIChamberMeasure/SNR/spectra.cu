#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"
#include <vector>
#include <sstream>
#include <stdexcept>
#include <iostream>

#define CUDA_ERROR_CHECK(ans) { cuda_assert_success((ans), __FILE__, __LINE__); }
#define CUFFT_ERROR_CHECK(ans) { cufft_assert_success((ans), __FILE__, __LINE__); }
#define LOG(str) { /*std::cout << (str) << std::endl;*/}
#define MAX_THREADS 1024

/**
 * @brief Function that raises an error on receipt of any cudaError_t
 *  value that is not cudaSuccess
 */
inline void cuda_assert_success(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
    {
      std::stringstream error_msg;
              error_msg << "CUDA failed with error: "
			<< hipGetErrorString(code) << std::endl
			<< "File: " << file << std::endl
			<< "Line: " << line << std::endl;
	      throw std::runtime_error(error_msg.str());
    }
}

inline void cufft_assert_success(hipfftResult code, const char *file, int line)
{
  if (code != HIPFFT_SUCCESS)
    {
      std::stringstream error_msg;
              error_msg << "CUFFT failed with error: "
			<< code << std::endl
			<< "File: " << file << std::endl
			<< "Line: " << line << std::endl;
	      throw std::runtime_error(error_msg.str());
    }
}

struct Plan
{
  hipfftHandle plan;
  hipfftComplex* input;
  hipfftComplex* output;
  float* detected;
  int size;
  int batch;
  int input_bytes;
  int output_bytes;
};

__global__
void detect_and_integrate(hipfftComplex* in, float* out, int nbins, int batch)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int size = nbins * batch;
  float xx = 0.0f, yy = 0.0f;
  for (int ii=idx; ii<size; ii+=nbins)
    {
      hipfftComplex val = in[ii];
      xx += val.x * val.x;
      yy += val.y * val.y;
    }
  out[idx] = xx + yy;
}

extern "C" {

  void init(Plan* plan)
  {
    CUDA_ERROR_CHECK(hipSetDevice(0));
    plan->input_bytes = (plan->size*plan->batch)*sizeof(hipfftComplex);
    plan->output_bytes = (plan->size)*sizeof(float);
    LOG("Device set");
    CUDA_ERROR_CHECK(hipMalloc(&(plan->input),plan->input_bytes));
    CUDA_ERROR_CHECK(hipMalloc(&(plan->output),plan->input_bytes));
    CUDA_ERROR_CHECK(hipMalloc(&(plan->detected),plan->output_bytes));
    LOG("All memory buffers generated");
    CUFFT_ERROR_CHECK(hipfftPlan1d(&(plan->plan),plan->size,HIPFFT_C2C,plan->batch));
    LOG("Plan generated");
    LOG(plan->input);
    LOG(plan->output);
    LOG(plan->detected);
  }

  void execute(Plan* plan, hipfftComplex* in, float* out)
  {
    int nthreads_per_block = std::min(plan->size, MAX_THREADS);
    int nblocks = plan->size / nthreads_per_block;
    LOG("Copy to device");
    LOG(plan->input);
    LOG(plan->output);
    LOG(plan->detected);
    LOG(in);
    LOG(out);
    CUDA_ERROR_CHECK(hipMemcpy(plan->input, in, plan->input_bytes, hipMemcpyHostToDevice));
    LOG("Done... executing FFT");
    CUFFT_ERROR_CHECK(hipfftExecC2C(plan->plan, plan->input, plan->output, HIPFFT_FORWARD));
    LOG("Done... detecting");
    detect_and_integrate<<<nblocks,nthreads_per_block>>>(plan->output, plan->detected, plan->size, plan->batch);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    LOG("Done... copying back to host");
    CUDA_ERROR_CHECK(hipMemcpy(out, plan->detected, plan->output_bytes, hipMemcpyDeviceToHost));
    LOG("Done.");
  }

  void deinit(Plan* plan)
  {
    CUDA_ERROR_CHECK(hipFree(plan->input));
    CUDA_ERROR_CHECK(hipFree(plan->output));
    CUDA_ERROR_CHECK(hipFree(plan->detected));
    CUFFT_ERROR_CHECK(hipfftDestroy(plan->plan));
  }

} //extern "C"

int main()
{
  Plan plan;
  plan.size = 523392;
  plan.batch = 256;
  hipfftComplex* input;
  float* output;
  CUDA_ERROR_CHECK(hipHostMalloc(&input,plan.size*plan.batch*sizeof(hipfftComplex), hipHostMallocDefault));
  CUDA_ERROR_CHECK(hipHostMalloc(&output,plan.size*sizeof(float), hipHostMallocDefault));
  init(&plan);
  execute(&plan,input,output);
  deinit(&plan);
  CUDA_ERROR_CHECK(hipHostFree(input));
  CUDA_ERROR_CHECK(hipHostFree(output));
}